#include "hip/hip_runtime.h"
﻿#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <ctype.h>


//Variables:

int** tablero;

//Funciones:

//Generación del tablero, el cual se encarga a la GPU para no sobrecargar la CPU:
__global__ void rellenarTablero(int* tablero, int nFilas, int nColumnas, int tiposN, hiprandState* state) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //Iniciar el generador aleatorio
    hiprand_init(3456, j, 0, &state[j]);
    if (j < nColumnas && i < nFilas && tablero[i * nColumnas + j] == 0) {
        tablero[i * nColumnas + j] = (hiprand(&state[i * nColumnas + j]) % tiposN + 1);
    }
}


//Comprueba que el bloque dado permita ser eliminado, y en caso afirmativo, elimina dichos elementos sobrescribiéndolos por 0:
__global__ void eliminarBloques(int* tablero, int nRows, int nColumns, int coordY, int coordX) {

    int fila = blockIdx.x;
    int columna = threadIdx.x;

    int carameloElegido = tablero[coordY * nRows + coordX];

    //Los hilos que pertenezcan a la fila de la posicion elegida ejecutan esto
    if (fila*columna < (nRows * nColumns) && fila == coordY ) {
        int start = coordX;
        int end = coordX;

        //Mientras haya caramelos iguales antes de nuestra posicion, llevar la posicion de la columna de inicio atras
        while (start > 0 && tablero[fila * nRows + start - 1] == carameloElegido) start--;

        //Mientras haya caramelos iguales despues de nuestra posicion, aumentar la posicion de la columna de fin.
        while (end < nColumns - 1 && tablero[fila * nRows + end + 1] == carameloElegido) end++;

        //Si la diferencia entre inicio y fin es mayor que 2, borramos todos los elementos poniendo un 0
        if (end - start + 1 >= 2) {
            for (int k = start; k <= end; k++) {
                tablero[fila * nRows + k] = 0;
            }
        }
    }

    __syncthreads();

    //Los hilos de la columna de la posicion elegida ejecutan el else:
    if (columna*fila < (nRows * nColumns) && columna == coordX) {
        int start = coordY;
        int end = coordY;

        //Igual que en el codigo de las filas, pero ahora vamos moviendo el inicio y final por las filas, en vez de las columnas
        while (start > 0 && tablero[(start - 1) * nRows + columna] == carameloElegido) start--;
        while (end < nRows - 1 && tablero[(end + 1) * nRows + columna] == carameloElegido) end++;
        //Remplazamos con 0s igual que en la fila
        if (end - start + 1 >= 2) {
            for (int k = start; k <= end; k++) {
                tablero[k * nRows + columna] = 0;
            }
        }
    }
}

//Eliminar el número de la fila o columna indicada por 'posActivar'. Si 'filaColumna' es True, entonces borra la fila, si es False, borra la columna:
__global__ void activarBomba(int* tablero, int posActivar, bool filaColumna, int nFilas, int nColumnas) {
    int x = threadIdx.x;
    //    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (filaColumna) {
        if (posActivar < nFilas) {
            if (x < nFilas) {
                tablero[posActivar * nColumnas + x] = 0;
            }
        }
        else {
            //De momento lo comento porque se imprime por cada hilo
            //printf("\n\n ERROR: No es posible borrar una fila fuera del rango de la matriz\n\n");
        }
    }
    else {
        if (posActivar < nColumnas) {
            if (x < nColumnas) {
                tablero[x * nColumnas + posActivar] = 0;
            }
        }
        else {
            //printf("\n\n ERROR: No es posible borrar una columna fuera del rango de la matriz\n\n");
        }
    }
}

//Eliminar todas las apariciones de un color de caramelo (que corresponde a un número entre 1-6) en el tablero:
__global__ void activarRompecabezas(int* tablero, int colorBloqueEliminar, int nFilas, int nColumnas) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i < nFilas * nColumnas) {
        //En caso de que la posición analizada sea igual al bloque que se quiere eliminar, se sobrescribe a 0
        if (tablero[i] == colorBloqueEliminar) {
            tablero[i] = 0;
        }
    }

}

//Eliminar todos los bloques en un radio de 4 elementos obteniendo como centro la posición indicada en las coordenadas ('posXActivar', 'posYActivar'):
__global__ void activarTNT(int* tablero, int posXActivar, int posYActivar, int nFilas, int nColumnas) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int radioExplosion = 2;     //Radio de bloques que afectará la explosión del TNT con respecto del centro, que es la posición introducida como parámetro de entrada

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i < nFilas * nColumnas) {
        //Comprobamos que la posición analizada esté dentro del rango de 'radioExplosion' elementos de radio
        if ((i >= posYActivar - radioExplosion) && (i <= posYActivar + radioExplosion) &&
            (j >= posXActivar - radioExplosion) && (j <= posXActivar + radioExplosion)) {
            tablero[i * nColumnas + j] = 0;
        }
    }
}

//Sobreescribir bloques con valor 0 con el valor de los bloques que se encuentren arriba de este. En caso de no tener bloques por encima, se generarán nuevos bloques:
__global__ void dejarCaerBloques(int* tablero, int nFilas, int nColumnas) {

    int i = threadIdx.x + blockIdx.x * blockDim.x; // nuevo índice que tiene en cuenta el número de hilos por bloque y bloques
    int posColumna = i % nColumnas;

    if (i<nFilas*nColumnas) {
        //Se recorre la columna en busca de algún 0:
        for (int lugarColumna = 0; lugarColumna < nFilas; ++lugarColumna) {
            if (tablero[posColumna + (nColumnas * lugarColumna)] == 0) {
                int posicionBloqueCero = posColumna + (nColumnas * lugarColumna);
                //En caso de encontrar un 0, vamos a iterar hasta que se encuentre en la primera fila de la matriz:
                while ((posicionBloqueCero / nColumnas) > 0) {
                    //printf("\nHilo %d Cambia su posicion %d por %d\n", i, tablero[posicionBloqueCero], tablero[posicionBloqueCero - nColumnas]);
                    tablero[posicionBloqueCero] = tablero[posicionBloqueCero - nColumnas];
                    tablero[posicionBloqueCero - nColumnas] = 0;
                    posicionBloqueCero -= nColumnas;
                }
                //Escribimos un 0 en la primera fila de la matriz:
                tablero[posicionBloqueCero] = 0;
            }
        }
    }
}

/*
__global__ void dejarCaerBloquees(int* tablero, int nFilas, int nColumnas) {

    int i = threadIdx.x; // calcula el índice correspondiente en la matriz
    int posColumna = i % nColumnas;
    int sigPosColumna = (i % nColumnas) + nColumnas;

    if (i<nFilas*nColumnas) {
        //Se recorre la columna en busca de algún 0:
        for (int lugarColumna = 0; lugarColumna < nFilas; ++lugarColumna) {
            if (tablero[posColumna + (nColumnas * lugarColumna)] == 0) {
                int posicionBloqueCero = posColumna + (nColumnas * lugarColumna);
                //printf("\nHilo %d Encuentra un 0 en %d. Dicho elemento es %d\n", i, posicionBloqueCero, tablero[posicionBloqueCero]);
                //En caso de encontrar un 0, vamos a iterar hasta que se encuentre en la primera fila de la matriz:
                while ((posicionBloqueCero / nColumnas) > 0) {
                    //printf("\nHilo %d entra en WHILE. Cambia %d por %d\n", i, tablero[posicionBloqueCero], tablero[posicionBloqueCero - nColumnas]);
                    tablero[posicionBloqueCero] = tablero[posicionBloqueCero - nColumnas];
                    tablero[posicionBloqueCero - nColumnas] = 0;
                    posicionBloqueCero -= nColumnas;
                }
                //Escribimos un 0 en la primera fila de la matriz:
                tablero[posicionBloqueCero] = 0;
            }
        }
    }
}*/


//-------------------------------------------------------------------------------------------------------------------------------------
// '1' si es fácil(1,2,3,4), '2' si es difícil(1,2,3,4,5,6) + número de filas del tablero + número de columnas del tablero


int validate_input(const char* prompt) {
    int num;
    char c;

    printf("%s", prompt);
    while (scanf("%d%c", &num, &c) != 2 || c != '\n') {
        while (getchar() != '\n');
        printf("Invalid input. %s", prompt);
    }

    return num;
}

int posicionesEliminadas(int* mtx, int m, int n) {
    int veces = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mtx[i * n + j] == 0) {
                veces++;
            }

        }
    }
    return veces;
}


//Impresión de la matriz por pantalla:
void print_matrix(int* mtx, int m, int n) {
    printf("\n");
    int valorCelda;
    for (int i = 0; i < m; i++) {
        printf("\t");
        for (int j = 0; j < n; j++) {
            valorCelda = mtx[i * n + j];
            if (valorCelda == 0) {
                //Si el valor es 0 (elemento borrado) no imprimimos nada
                printf("   ");
            }
            else if (valorCelda == 10) {
                //La bomba se representa con B al imprimir
                printf(" B ");
            }
            else if (valorCelda == 20) {
                //La TNT se representa con T al imprimir
                printf(" T ");
            }
            else if (valorCelda > 49 && valorCelda < 57) {
                //El rompecabezas se representa con Rx al imprimir
                printf("R%d ", (valorCelda % 10));
            }
            else {
                //Imprimimos el valor del caramelo
                printf(" %d ", valorCelda);
            }
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char** argv) {
    const int filas = 10;
    const int columnas = 10;
    int tiposCaramelos = 4;
    int vidas = 5;
    int modo = 1; //1 manual, 2 automatico

    int* tablero_dev;
    int tablero_host[filas][columnas];

    //Llenar de 0 la matriz inicial
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            tablero_host[i][j] = 0;
        }
    }

    hiprandState* state;

    //Dar memoria a la matriz y el generador aleatorio en la GPU
    hipMalloc((void**)&state, filas * columnas * sizeof(hiprandState));
    hipMalloc((void**)&tablero_dev, filas * columnas * sizeof(int));

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    dim3 blocks(filas, columnas);
    dim3 threads(filas, columnas);
    printf("\nGeneracion inicial del tablero:\n");


    //BUCLE DEL JUEGO!!!
    int coordX;
    int coordY;
    /*
    while (true) {
        rellenarTablero << < blocks, threads >> > (tablero_dev, filas, columnas, tiposCaramelos, state);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        tablero_host[8][7] = 0;
        tablero_host[9][7] = 0;
        tablero_host[6][7] = 0;
        tablero_host[7][7] = 0;
        tablero_host[5][7] = 0;
        tablero_host[4][7] = 20;
        printf("Imprimimos matriz\n");
        print_matrix((int*)tablero_host, filas, columnas);
        getchar();
        printf("Dejamos caer elementos\n");
        hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
        dejarCaerBloquees << <blocks, threads >> > (tablero_dev, filas, columnas);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        printf("Imprimimos matriz 222222\n");
        print_matrix((int*)tablero_host, filas, columnas);
        getchar();
        hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
        rellenarTablero << < blocks, threads >> > (tablero_dev, filas, columnas, tiposCaramelos, state);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        printf("Imprimimos matriz 333333\n");
        print_matrix((int*)tablero_host, filas, columnas);
        getchar();

    }*/


    while (vidas > 0) {
        //Al empezar cada ronda, rellenar el tablero con caramelos
        system("cls");
        printf("\n \t\tCUNDY CROSH SOGA\n");
        printf("----------------------------------------------------------------\n");
        printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
        printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
        rellenarTablero << < blocks, threads >> > (tablero_dev, filas, columnas, tiposCaramelos, state);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        print_matrix((int*)tablero_host, filas, columnas);

        //Pedir las coordenadas al usuario
        if (modo == 1) {
            coordY = validate_input("Introduce la coordenada Y (fila): ") - 1;
            coordX = validate_input("Introduce la coordenada X (columna): ") - 1;
        }
        else {
            coordY = rand() % filas;
            coordX = rand() % columnas;
            printf("Posicion elegida aleatoriamente: Fila %d, Columna %d", coordY + 1, coordX + 1);
            getchar();
        }
        
        //Intentar eliminar bloques en la posicion que se ha indicado
        if (tablero_host[coordY][coordX] == 10) {
            bool filaCol = rand() % 2;
            if (filaCol) {
                activarBomba << <blocks, threads >> > (tablero_dev, coordY, filaCol, filas, columnas);
            }
            else if (filaCol) {
                activarBomba << <blocks, threads >> > (tablero_dev, coordX, filaCol, filas, columnas);
            }
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        }
        else if (tablero_host[coordY][coordX] == 20) {
            activarTNT << <blocks, threads >> > (tablero_dev, coordX, coordY, filas, columnas);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        }
        else if (tablero_host[coordY][coordX] > 49 && tablero_host[coordY][coordX] < 57) {
            activarRompecabezas << <blocks,threads >> > (tablero_dev, tablero_host[coordY][coordX]%10, filas, columnas);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        }
        else {
            eliminarBloques << < filas, columnas>> > (tablero_dev, filas, columnas, coordY, coordX);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
            int posiciones = posicionesEliminadas((int*)tablero_host, filas, columnas);
            if (posiciones == 5) {
                //El 10 es una bomba
                tablero_host[coordY][coordX] = 10;
            }
            else if (posiciones == 6) {
                //El 20 es una TNT
                tablero_host[coordY][coordX] = 20;
            }
            else if (posiciones > 6) {
                //El 5x es un rompecabezas
                tablero_host[coordY][coordX] = 50 + tablero_host[coordY][coordX]%10;
            }

        }
 
        if (posicionesEliminadas((int*)tablero_host, filas, columnas) == 0) {
            //Si no se ha eliminado ningun caramelo con el kernel
            vidas--;
            printf("\nPosicion mala: te quedan %d vidas\n", vidas);
            getchar();
        }
        else {
            //Cuando si se ha modificado el tablero
            system("cls");
            printf("\n \t\tCUNDY CROSH SOGA\n");
            printf("----------------------------------------------------------------\n");
            printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
            printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
            system("cls");
            printf("\n \t\tCUNDY CROSH SOGA\n");
            printf("----------------------------------------------------------------\n");
            printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
            printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
            dejarCaerBloques << <blocks, columnas>> > (tablero_dev, filas, columnas);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
        }
    }

    printf("\n\tGAME OVER X_X\n");
    printf("\n\tGracias por jugar!\n");
    printf("\n\tBy: Daniel De Heras y Adrian Borges\n");
    printf("\n\n-------------------------------------------------------\n\n");

    hipFree(tablero_dev);
    hipFree(state);

    return 0;
}


