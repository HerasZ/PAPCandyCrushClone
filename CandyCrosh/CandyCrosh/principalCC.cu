﻿
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <time.h>
#include <stdio.h>

//Dividir qué funciones se ejecutarán en la GPU y qué funciones en la CPU:

//Elementos:

int** tablero;
const int numVidas = 3;

//Funciones:

//Generación del tablero, el cual se encarga a la GPU para no sobrecargar la CPU:
__global__ void generarTablero(int* tablero, int nFilas, int nColumnas, int tiposN, hiprandState* state) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    //Iniciar el generador aleatorio
    hiprand_init(3456, j, 0, &state[j]);
    if (j < nColumnas && i < nFilas) {
        tablero[i * nColumnas + j] = (hiprand(&state[i * nColumnas + j])%tiposN+1);
    }
}

void print_matrix(int* mtx, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", mtx[i*n+j]);
        }
        printf("\n");
    }
}

//-------------------------------------------------------------------------------------------------------------------------------------
// '1' si es fácil(1,2,3,4), '2' si es difícil(1,2,3,4,5,6) + número de filas del tablero + número de columnas del tablero

int main(int argc, char** argv) { 
    const int filas = 8;
    const int columnas = 8;
    int tiposCaramelos = 6;

    int* tablero_dev;
    int tablero_host[filas][columnas];

    //Llenar de 0 la matriz inicial
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            tablero_host[i][j] = 0;
        }
    }
    print_matrix((int*)tablero_host, filas, columnas);

    hiprandState* state;

    //Dar memoria a la matriz y el generador aleatorio en la GPU
    hipMalloc((void**)&state, filas * columnas * sizeof(hiprandState));
    hipMalloc((void**)&tablero_dev,filas*columnas*sizeof(int));

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks(4, 4);
    dim3 threads(16, 16);
    generarTablero<<< blocks,threads >>>(tablero_dev,filas,columnas,tiposCaramelos,state);

    hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);

    printf("\n");
    print_matrix((int*)tablero_host, filas, columnas);

    return 0;
}





