#include "hip/hip_runtime.h"
﻿
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <ctype.h>


//Variables:

int** tablero;

//Funciones:

//Generación del tablero, el cual se encarga a la GPU para no sobrecargar la CPU:
__global__ void rellenarTablero(int* tablero, int nFilas, int nColumnas, int tiposN, hiprandState* state) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //Iniciar el generador aleatorio
    hiprand_init(3456, j, 0, &state[j]);
    if (j < nColumnas && i < nFilas && tablero[i * nColumnas + j]==0) {
        tablero[i * nColumnas + j] = (hiprand(&state[i * nColumnas + j])%tiposN+1);
    }
}


//Comprueba que el bloque dado permita ser eliminado, y en caso afirmativo, elimina dichos elementos sobrescribiéndolos por 0:
__global__ void eliminarBloques(int* tablero, int nRows, int nColumns, int coordY, int coordX) {
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int carameloElegido = tablero[coordY * nRows + coordX];
    __shared__ int posicionesEliminadas;   

    //Los hilos que pertenezcan a la fila de la posicion elegida ejecutan esto
    if (tid < (nRows*nColumns) && tid == coordY) {
        int start = coordX;
        int end = coordX;
        
        //Mientras haya caramelos iguales antes de nuestra posicion, llevar la posicion de la columna de inicio atras
        while (start > 0 && tablero[tid * nRows + start - 1] == carameloElegido) start--;

        //Mientras haya caramelos iguales despues de nuestra posicion, aumentar la posicion de la columna de fin.
        while (end < nColumns - 1 && tablero[tid * nRows + end + 1] == carameloElegido) end++;

        //Si la diferencia entre inicio y fin es mayor que 2, borramos todos los elementos poniendo un 0
        if (end - start + 1 >= 2) {
            for (int i = start; i <= end; i++) {
                tablero[tid * nRows + i] = 0;
                atomicAdd(&posicionesEliminadas,1);
            }
        }
    }
    //Los hilos de la columna de la posicion elegida ejecutan el else:
    else if (tid < (nRows*nColumns) && tid == coordX) {
        int start = coordY;
        int end = coordY;
        //Igual que en el codigo de las filas, pero ahora vamos moviendo el inicio y final por las filas, en vez de las columnas
        while (start > 0 && tablero[(start - 1) * nRows + tid] == carameloElegido) start--;
        while (end < nRows - 1 && tablero[(end + 1) * nRows + tid] == carameloElegido) end++;
        //Remplazamos con 0s igual que en la fila
        if (end - start + 1 >= 2) {
            for (int i = start; i <= end; i++) {
                tablero[i * nRows + tid] = 0;
                atomicAdd(&posicionesEliminadas, 1);
            }
        }
    }
    __syncthreads();

    if (posicionesEliminadas == 5) {
        //El 10 es una bomba
        tablero[coordY * nRows + coordX] = 10;
    }
    else if (posicionesEliminadas == 6) {
        //El 20 es una TNT
        tablero[coordY * nRows + coordX] = 20;
    }
    else if (posicionesEliminadas > 6) {
        //El 33 es un rompecabezas
        tablero[coordY * nRows + coordX] = 50 + carameloElegido;
    }
}

//Eliminar el número de la fila o columna indicada por 'posActivar'. Si 'filaColumna' es True, entonces borra la fila, si es False, borra la columna:
__global__ void activarBomba(int* tablero, int posActivar, bool filaColumna, int nFilas, int nColumnas ) {
    int x = threadIdx.x;
//    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (filaColumna) {
        if (posActivar < nFilas) {
            if (x < nFilas) {
                tablero[posActivar * nColumnas + x] = 0;
        }}
        else {
            //De momento lo comento porque se imprime por cada hilo
            //printf("\n\n ERROR: No es posible borrar una fila fuera del rango de la matriz\n\n");
    }}
    else {
        if (posActivar < nColumnas) {
            if (x < nColumnas) {
                tablero[x * nColumnas + posActivar] = 0;
            }} else{
            //printf("\n\n ERROR: No es posible borrar una columna fuera del rango de la matriz\n\n");
    }} 
}

//Eliminar todas las apariciones de un color de caramelo (que corresponde a un número entre 1-6) en el tablero:
__global__ void activarRompecabezas(int* tablero, int colorBloqueEliminar, int nFilas, int nColumnas) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i < nFilas*nColumnas) {     
        //En caso de que la posición analizada sea igual al bloque que se quiere eliminar, se sobrescribe a 0
        if (tablero[i] == colorBloqueEliminar) {
            tablero[i] = 0;
        }
    }
    
}

//Eliminar todos los bloques en un radio de 4 elementos obteniendo como centro la posición indicada en las coordenadas ('posXActivar', 'posYActivar'):
__global__ void activarTNT(int* tablero, int posXActivar, int posYActivar, int nFilas, int nColumnas) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int radioExplosion = 2;     //Radio de bloques que afectará la explosión del TNT con respecto del centro, que es la posición introducida como parámetro de entrada

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i < nFilas * nColumnas) {
        //Comprobamos que la posición analizada esté dentro del rango de 'radioExplosion' elementos de radio
        if ((i >= posYActivar - radioExplosion) && (i <= posYActivar + radioExplosion) &&
            (j >= posXActivar - radioExplosion) && (j <= posXActivar + radioExplosion)){
                tablero[i*nColumnas+j] = 0;
        }
    }
}

//Sobreescribir bloques con valor 0 con el valor de los bloques que se encuentren arriba de este. En caso de no tener bloques por encima, se generarán nuevos bloques:
//*PONER EN LA MEMORIA QUE TBN SE ME HABIA OCURRIDO HACER QUE SE SUBA EL 0 Y BAJAR UNA LSITA CON EL RESTO DE ELEMENTOS, PERO COMO ES COMPLICADO TRABAJAR CON ARRAYS DINAMICOS, FUE DESCARTADO
__global__ void dejarCaerBloques(int* tablero, int nFilas, int nColumnas) {
    
    int i = threadIdx.x; // calcula el índice correspondiente en la matriz
    int posColumna = i % nColumnas;
    int sigPosColumna = (i % nColumnas) + nColumnas;

    //Se recorre la columna en busca de algún 0:
    for (int lugarColumna = 0; lugarColumna < nFilas; ++lugarColumna) {
        if (tablero[posColumna + (nColumnas * lugarColumna)] == 0) {
            int posicionBloqueCero = posColumna + (nColumnas * lugarColumna);
            //En caso de encontrar un 0, vamos a iterar hasta que se encuentre en la primera fila de la matriz:
            while ((posicionBloqueCero / nColumnas) > 0) {
                tablero[posicionBloqueCero] = tablero[posicionBloqueCero-nColumnas];
                tablero[posicionBloqueCero - nColumnas] = 0;
                posicionBloqueCero -= nColumnas;
            }
            //Escribimos un 0 en la primera fila de la matriz:
            tablero[posicionBloqueCero] = 0;
        }
    }
}


//-------------------------------------------------------------------------------------------------------------------------------------
// '1' si es fácil(1,2,3,4), '2' si es difícil(1,2,3,4,5,6) + número de filas del tablero + número de columnas del tablero


int validate_input(const char* prompt) {
    int num;
    char c;

    printf("%s", prompt);
    while (scanf("%d%c", &num, &c) != 2 || c != '\n') {
        while (getchar() != '\n');
        printf("Invalid input. %s", prompt);
    }

    return num;
}

int posicionesEliminadas(int* mtx, int m, int n) {
    int veces = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mtx[i * n + j] == 0) {
                veces++;
            }
            
        }
    }
    return veces;
}


//Impresión de la matriz por pantalla:
void print_matrix(int* mtx, int m, int n) {
    int valorCelda;
    for (int i = 0; i < m; i++) {
        printf("\t");
        for (int j = 0; j < n; j++) {
            valorCelda = mtx[i * n + j];
            if (valorCelda == 0) {
                //Si el valor es 0 (elemento borrado) no imprimimos nada
                printf("  ");
            }
            else if(valorCelda == 10) {
                //La bomba se representa con B al imprimir
                printf("B ");
            }
            else if (valorCelda == 20) {
                //La TNT se representa con T al imprimir
                printf("T ");
            }
            else if (valorCelda > 49 && valorCelda < 57) {
                //El rompecabezas se representa con Rx al imprimir
                printf("R%d",(valorCelda%10));
            }
            else {
                //Imprimimos el valor del caramelo
                printf("%d ", valorCelda);
            }
        }
        printf("\n");
    }
}

int main(int argc, char** argv) { 
    const int filas = 10; 
    const int columnas = 10;
    int tiposCaramelos = 4;
    int vidas = 5;

    int* tablero_dev;
    int tablero_host[filas][columnas];

    //Llenar de 0 la matriz inicial
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            tablero_host[i][j] = 0;
        }
    }

    hiprandState* state;

    //Dar memoria a la matriz y el generador aleatorio en la GPU
    hipMalloc((void**)&state, filas * columnas * sizeof(hiprandState));
    hipMalloc((void**)&tablero_dev,filas*columnas*sizeof(int));

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    dim3 blocks(filas, columnas);
    dim3 threads(filas, columnas);
    printf("\nGeneracion inicial del tablero:\n");
    

    //BUCLE DEL JUEGO!!!
    int coordX;
    int coordY;

    while (vidas > 0) {
        //Al empezar cada ronda, rellenar el tablero con caramelos
        system("cls");
        rellenarTablero << < blocks, threads >> > (tablero_dev, filas, columnas, tiposCaramelos, state);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        print_matrix((int*)tablero_host, filas, columnas);

        //Pedir las coordenadas al usuario
        coordY = validate_input("Introduce la coordenada Y (fila): ") - 1;
        coordX = validate_input("Introduce la coordenada X (columna): ") - 1;
        

        //Intentar eliminar bloques en la posicion que se ha indicado

        //TODO: Comprobar si la posicion que hemos elegido es un caramelo, rompecabezas, o distintos para ejecutar 
        // el kernel que corresponde
        eliminarBloques << <1, filas+columnas >> > (tablero_dev, filas, columnas, coordY, coordX);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);


        if (posicionesEliminadas((int*)tablero_host,filas,columnas) == 0) {
            //Si no se ha eliminado ningun caramelo con el kernel
            vidas--;
            printf("\nPosicion mala: te quedan %d vidas\n", vidas);
            getchar();
        }
        else {
            //Cuando si se ha modificado el tablero
            system("cls");
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
            system("cls");
            dejarCaerBloques << <1, columnas >> > (tablero_dev, filas, columnas);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
        }
    }
    
    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    //activarBomba << <blocks, threads >> > (tablero_dev, 2, 1, filas, columnas);          //Se deben mandar los hilos equivalentes a la longitud de la fila
    printf("\nActivacion del TNT en (4,5):\n");
    activarTNT << <blocks, threads >> > (tablero_dev, 4,5, filas, columnas);
    //printf("\nActivacion del rompecabezas con el numero 4:\n");
    //activarRompecabezas << <blocks,threads >> > (tablero_dev, 4, filas, columnas);     //Se deben lanzar los hilos equivalentes al tamaño de la matriz
    //eliminarBloques << <1, filas*columnas >> > (tablero_dev, filas, 2, 2);
    hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
    printf("\n");
    print_matrix((int*)tablero_host, filas, columnas);

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    printf("\nDejar caer bloques por la gravedad, subiendo los ceros:\n");
    dejarCaerBloques << <1, columnas >> > (tablero_dev, filas, columnas);     //Se deben lanzar los hilos equivalentes al número de columnas
    hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
    printf("\n");
    print_matrix((int*)tablero_host, filas, columnas);

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    printf("\nSobrescribir los ceros del tablero por nuevos numeros generados aleatoriamente:\n");
    rellenarTablero << <1, threads >> > (tablero_dev, filas, columnas,tiposCaramelos, state);     //Se deben lanzar los hilos equivalentes al número de columnas
    hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
    printf("\n");
    print_matrix((int*)tablero_host, filas, columnas);


    hipFree(tablero_dev);

    return 0;
}





