#include "hip/hip_runtime.h"
﻿#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <ctype.h>


//Variables:

int** tablero;

//Funciones:

//Generación del tablero, el cual se encarga a la GPU para no sobrecargar la CPU:
__global__ void rellenarTablero(int* tablero, int nFilas, int nColumnas, int tiposN, hiprandState* state) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //Iniciar el generador aleatorio
    hiprand_init(3456, j, 0, &state[j]);
    if (j < nColumnas && i < nFilas && tablero[i * nColumnas + j] == 0) {
        tablero[i * nColumnas + j] = (hiprand(&state[i * nColumnas + j]) % tiposN + 1);
    }
}


//Comprueba que el bloque dado permita ser eliminado, y en caso afirmativo, elimina dichos elementos sobrescribiéndolos por 0:
__global__ void eliminarBloques(int* tablero, int nRows, int nColumns, int coordY, int coordX) {

    __shared__ int matrizCompartida[10][10];

    int fila = blockIdx.x * blockDim.x + threadIdx.x;
    int columna = blockIdx.y * blockDim.y + threadIdx.y;

    if (fila < nColumns && columna < nRows) {
        matrizCompartida[fila][columna] = tablero[fila * nRows + columna];
    }

    __syncthreads();

    int carameloElegido = tablero[coordY * nRows + coordX];

    //Los hilos que pertenezcan a la fila de la posicion elegida ejecutan esto
    if (fila < nColumns && columna < nRows && fila == coordY) {
        int start = coordX;
        int end = coordX;

        //Mientras haya caramelos iguales antes de nuestra posicion, llevar la posicion de la columna de inicio atras
        while (start > 0 && matrizCompartida[fila][start-1] == carameloElegido) start--;

        //Mientras haya caramelos iguales despues de nuestra posicion, aumentar la posicion de la columna de fin.
        while (end < nColumns - 1 && matrizCompartida[fila][end + 1] == carameloElegido) end++;

        //Si la diferencia entre inicio y fin es mayor que 2, borramos todos los elementos poniendo un 0
        if (end - start + 1 >= 2) {
            for (int k = start; k <= end; k++) {
                tablero[fila * nRows + k] = 0;
            }
        }
    }

    //Los hilos de la columna de la posicion elegida ejecutan el else:
    if (fila < nColumns && columna < nRows && columna == coordX) {
        int start = coordY;
        int end = coordY;

        //Igual que en el codigo de las filas, pero ahora vamos moviendo el inicio y final por las filas, en vez de las columnas
        while (start > 0 && matrizCompartida[start - 1][columna] == carameloElegido) start--;
        while (end < nRows - 1 && matrizCompartida[end + 1][columna] == carameloElegido) end++;
        //Remplazamos con 0s igual que en la fila
        if (end - start + 1 >= 2) {
            for (int k = start; k <= end; k++) {
                tablero[k * nRows + columna] = 0;
            }
        }
    }
}

//Eliminar el número de la fila o columna indicada por 'posActivar'. Si 'filaColumna' es True, entonces borra la fila, si es False, borra la columna:
__global__ void activarBomba(int* tablero, int posActivar, bool filaColumna, int nFilas, int nColumnas) {
    int x = threadIdx.x;
    //    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (filaColumna) {
        if (posActivar < nFilas) {
            if (x < nFilas) {
                tablero[posActivar * nColumnas + x] = 0;
            }
        }
        else {
            //De momento lo comento porque se imprime por cada hilo
            //printf("\n\n ERROR: No es posible borrar una fila fuera del rango de la matriz\n\n");
        }
    }
    else {
        if (posActivar < nColumnas) {
            if (x < nColumnas) {
                tablero[x * nColumnas + posActivar] = 0;
            }
        }
        else {
            //printf("\n\n ERROR: No es posible borrar una columna fuera del rango de la matriz\n\n");
        }
    }
}

//Eliminar todas las apariciones de un color de caramelo (que corresponde a un número entre 1-6) en el tablero:
__global__ void activarRompecabezas(int* tablero, int colorBloqueEliminar, int nFilas, int nColumnas, int coordX, int coordY) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i*j < nFilas * nColumnas) {
        //En caso de que la posición analizada sea igual al bloque que se quiere eliminar, se sobrescribe a 0
        if (tablero[i * nFilas + j] == colorBloqueEliminar) {
            tablero[i * nFilas + j] = 0;
        }
    }
    tablero[coordY * nFilas + coordX] = 0;

}

//Eliminar todos los bloques en un radio de 4 elementos obteniendo como centro la posición indicada en las coordenadas ('posXActivar', 'posYActivar'):
__global__ void activarTNT(int* tablero, int posXActivar, int posYActivar, int nFilas, int nColumnas) { //'nColumnas' como parámetro para asegurarse de recorrer y borrar todas las apariciones en la matriz
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int radioExplosion = 2;     //Radio de bloques que afectará la explosión del TNT con respecto del centro, que es la posición introducida como parámetro de entrada

    //Comprobamos que el índice se encuentre dentro de los límites de la matriz
    if (i < nFilas * nColumnas) {
        //Comprobamos que la posición analizada esté dentro del rango de 'radioExplosion' elementos de radio
        if ((i >= posYActivar - radioExplosion) && (i <= posYActivar + radioExplosion) &&
            (j >= posXActivar - radioExplosion) && (j <= posXActivar + radioExplosion)) {
            tablero[i * nColumnas + j] = 0;
        }
    }
}

//Sobreescribir bloques con valor 0 con el valor de los bloques que se encuentren arriba de este. En caso de no tener bloques por encima, se generarán nuevos bloques:
__global__ void dejarCaerBloques(int* tablero, int nFilas, int nColumnas) {

    // nuevo índice que tiene en cuenta el número de hilos por bloque y bloques
    int i = blockIdx.x;
    int posColumna = threadIdx.x;

    if (i * posColumna < nFilas * nColumnas) {
        //Se recorre la columna en busca de algún 0:
        for (int lugarColumna = 0; lugarColumna < nFilas; ++lugarColumna) {
            if (tablero[posColumna + (nColumnas * lugarColumna)] == 0) {
                int posicionBloqueCero = posColumna + (nColumnas * lugarColumna);
                //En caso de encontrar un 0, vamos a iterar hasta que se encuentre en la primera fila de la matriz:
                while ((posicionBloqueCero) >= posColumna) {
                    //printf("\nHilo %d Cambia su posicion %d por %d\n", i, tablero[posicionBloqueCero], tablero[posicionBloqueCero - nColumnas]);
                    tablero[posicionBloqueCero] = tablero[posicionBloqueCero - nColumnas];
                    tablero[posicionBloqueCero - nColumnas] = 0;
                    posicionBloqueCero -= nColumnas;
                }
            }
        }
    }
}

__device__ int posicionesCero = 0;
__global__ void ponerPowerup(int* tablero, int nFilas, int nColumnas, int coordY, int coordX, int carameloEnPos) {
    int fila = blockIdx.x;
    int columna = threadIdx.x;
    posicionesCero = 0;
    if (tablero[fila * nFilas + columna] == 0) {
        atomicAdd(&posicionesCero, 1);
    }
    __syncthreads();
    if (posicionesCero == 5) {
        //El 10 es una bomba
        tablero[coordY * nFilas + coordX] = 10;
    }
    else if (posicionesCero == 6) {
        //El 20 es una TNT
        tablero[coordY * nFilas + coordX] = 20;
    }
    else if (posicionesCero > 6) {
        //El 5x es un rompecabezas
        tablero[coordY * nFilas + coordX] = 50 + carameloEnPos % 10;
    }
}


//-------------------------------------------------------------------------------------------------------------------------------------
// '1' si es fácil(1,2,3,4), '2' si es difícil(1,2,3,4,5,6) + número de filas del tablero + número de columnas del tablero


int validate_input(const char* prompt) {
    int num;
    char c;

    printf("%s", prompt);
    while (scanf("%d%c", &num, &c) != 2 || c != '\n') {
        while (getchar() != '\n');
        printf("Invalid input. %s", prompt);
    }

    return num;
}

int posicionesEliminadas(int* mtx, int m, int n) {
    int veces = 0;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mtx[i * n + j] == 0) {
                veces++;
            }

        }
    }
    return veces;
}


//Impresión de la matriz por pantalla:
void print_matrix(int* mtx, int m, int n) {
    printf("\n");
    int valorCelda;
    for (int i = 0; i < m; i++) {
        printf("\t");
        for (int j = 0; j < n; j++) {
            valorCelda = mtx[i * n + j];
            if (valorCelda == 0) {
                //Si el valor es 0 (elemento borrado) no imprimimos nada
                printf("   ");
            }
            else if (valorCelda == 10) {
                //La bomba se representa con B al imprimir
                printf(" B ");
            }
            else if (valorCelda == 20) {
                //La TNT se representa con T al imprimir
                printf(" T ");
            }
            else if (valorCelda > 49 && valorCelda < 57) {
                //El rompecabezas se representa con Rx al imprimir
                printf("R%d ", (valorCelda % 10));
            }
            else {
                //Imprimimos el valor del caramelo
                printf(" %d ", valorCelda);
            }
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char** argv) {
    const int filas = 10;
    const int columnas = 10;
    int tiposCaramelos = 4;
    int vidas = 5;
    int modo = 1; //1 manual, 2 automatico

    int* tablero_dev;
    int tablero_host[filas][columnas];

    //Llenar de 0 la matriz inicial
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            tablero_host[i][j] = 0;
        }
    }

    hiprandState* state;

    //Dar memoria a la matriz y el generador aleatorio en la GPU
    hipMalloc((void**)&state, filas * columnas * sizeof(hiprandState));
    hipMalloc((void**)&tablero_dev, filas * columnas * sizeof(int));

    hipMemcpy(tablero_dev, tablero_host, filas * columnas * sizeof(int), hipMemcpyHostToDevice);
    dim3 blocks(filas, columnas);
    dim3 threads(filas, columnas);
    printf("\nGeneracion inicial del tablero:\n");


    //BUCLE DEL JUEGO!!!
    int coordX;
    int coordY;


    while (vidas > 0) {
        //Al empezar cada ronda, rellenar el tablero con caramelos
        system("cls");
        printf("\n \t\tCUNDY CROSH SOGA\n");
        printf("----------------------------------------------------------------\n");
        printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
        printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
        rellenarTablero << < blocks, threads >> > (tablero_dev, filas, columnas, tiposCaramelos, state);
        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
        print_matrix((int*)tablero_host, filas, columnas);

        //Pedir las coordenadas al usuario
        if (modo == 1) {
            coordY = validate_input("Introduce la coordenada Y (fila): ") - 1;
            coordX = validate_input("Introduce la coordenada X (columna): ") - 1;
        }
        else {
            coordY = rand() % filas;
            coordX = rand() % columnas;
            printf("Posicion elegida aleatoriamente: Fila %d, Columna %d", coordY + 1, coordX + 1);
            getchar();
        }

        int valor = tablero_host[coordY][coordX];

        //Intentar eliminar bloques en la posicion que se ha indicado
        if (tablero_host[coordY][coordX] == 10) {
            bool filaCol = rand() % 2;
            if (filaCol) {
                activarBomba << <blocks, threads >> > (tablero_dev, coordY, filaCol, filas, columnas);
            }
            else if (filaCol) {
                activarBomba << <blocks, threads >> > (tablero_dev, coordX, filaCol, filas, columnas);
            }
        }
        else if (tablero_host[coordY][coordX] == 20) {
            activarTNT << <blocks, threads >> > (tablero_dev, coordX, coordY, filas, columnas);
        }
        else if (tablero_host[coordY][coordX] > 49 && tablero_host[coordY][coordX] < 57) {
            activarRompecabezas << <blocks, threads >> > (tablero_dev, tablero_host[coordY][coordX] % 10, filas, columnas, coordX, coordY);
        }
        else {
            
            eliminarBloques << < blocks, threads >> > (tablero_dev, filas, columnas, coordY, coordX);
            ponerPowerup << <filas, columnas >> > (tablero_dev, filas, columnas, coordY, coordX, valor);
        }

        hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);

        if (posicionesEliminadas((int*)tablero_host, filas, columnas) == 0) {
            //Si no se ha eliminado ningun caramelo con el kernel
            vidas--;
            printf("\nPosicion mala: te quedan %d vidas\n", vidas);
            getchar();
        }
        else {
            //Cuando si se ha modificado el tablero
            system("cls");
            printf("\n \t\tCUNDY CROSH SOGA\n");
            printf("----------------------------------------------------------------\n");
            printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
            printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
            system("cls");
            printf("\n \t\tCUNDY CROSH SOGA\n");
            printf("----------------------------------------------------------------\n");
            printf("*Paradigmas Avanzados de Programacion, 3GII* 31 de marzo de 2023\n");
            printf("By: Daniel de Heras Zorita y Adrian Borges Cano\n");
            dejarCaerBloques << <filas, columnas >> > (tablero_dev, filas, columnas);
            hipMemcpy(tablero_host, tablero_dev, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);
            print_matrix((int*)tablero_host, filas, columnas);
            getchar();
        }
    }

    printf("\n\tGAME OVER X_X\n");
    printf("\n\tGracias por jugar!\n");
    printf("\n\tBy: Daniel De Heras y Adrian Borges\n");
    printf("\n\n-------------------------------------------------------\n\n");

    hipFree(tablero_dev);
    hipFree(state);

    return 0;
}


